//
// Created by Jan Warchocki on 06/03/2022.
//

#include "verify.cuh"
#include <iostream>

#ifdef HAS_CUDA

#include "hip/hip_runtime.h"

bool isCudaAvailable() {
    return true;
}

void showCudaInfo() {
    // Borrowed from https://stackoverflow.com/questions/5689028
    int kb = 1024;
    int mb = kb * kb;

    std::cout << "CUDA VERSION v" << CUDA_VERSION << std::endl;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "Detected " << deviceCount << (deviceCount == 1 ? " device:" : " devices:") << std::endl;

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        std::cout << "DEVICE " << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
        std::cout << "\tGlobal memory:   " << props.totalGlobalMem / mb << "MB" << std::endl;
        std::cout << "\tShared memory:   " << props.sharedMemPerBlock / kb << "KB" << std::endl;
        std::cout << "\tConstant memory: " << props.totalConstMem / kb << "KB" << std::endl;
        std::cout << "\tBlock registers: " << props.regsPerBlock << std::endl;

        std::cout << "\tWarp size:         " << props.warpSize << std::endl;
        std::cout << "\tThreads per block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "\tMax block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
        std::cout << "\tMax grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << std::endl;
    }

    std::cout << std::endl;
}

#else

bool isCudaAvailable() {
    return false;
}

void showCudaInfo() {
    std::cout << "No version of CUDA is available." << std::endl << std::endl;
}

#endif