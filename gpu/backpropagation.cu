#include "hip/hip_runtime.h"
//
// Created by Jan Warchocki on 10/03/2022.
//

#include "backpropagation.cuh"
#include "verify.cuh"
#include "allocation_gpu.cuh"
#include "assert.cuh"

#ifdef HAS_CUDA

#include <hip/hip_runtime.h>
#define NUM_BLOCKS 200

__global__
void performBackpropagation(DTYPE* biases, DTYPE* weights, DTYPE* data, DTYPE* derivatives,
                            DTYPE* delta, DTYPE* previousWeights, DTYPE* newDelta, int inSize, int outSize, int deltaSize,
                            DTYPE learningRate, bool isLastLayer) {
    auto index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= outSize) {
        return;
    }

    if (isLastLayer) {
        DTYPE coreGradient = delta[index] * derivatives[index];

        biases[index] -= learningRate * coreGradient;

        for (int j = 0; j < inSize; j++) {
            weights[index * inSize + j] -= learningRate * coreGradient * data[j];
        }

        newDelta[index] = coreGradient;
    } else {
        DTYPE coreGradient = 0;
        for (int j = 0; j < deltaSize; j++) {
            coreGradient += delta[j] * derivatives[index] * previousWeights[j * outSize + index];
        }

        biases[index] -= learningRate * coreGradient;

        for (int j = 0; j < inSize; j++) {
            weights[index * inSize + j] -= learningRate * coreGradient * data[j];
        }

        newDelta[index] = coreGradient;
    }
}

Vector backpropagation(Layer& layer, const Vector& delta, const Matrix& previousWeights,
              bool isLastLayer, DTYPE learningRate) {
    Vector derivatives = layer.calculateDerivatives();
    derivatives.moveToDevice();


    Vector newDelta = Vector(layer.outSize, DEVICE);

    performBackpropagation<<<1, layer.outSize>>>(layer.biases.data, layer.weights.data,
                                                 layer.data.data, derivatives.data, delta.data,
                                                 previousWeights.data, newDelta.data,
                                                 layer.inSize, layer.outSize, delta.n, learningRate, isLastLayer);

    return newDelta;
}

#else

Vector backpropagation(Layer& layer, const Vector& delta, const Matrix& previousWeights,
              bool isLastLayer, DTYPE learningRate) {
    const Vector& derivatives = layer.calculateDerivatives();
    if (!isLastLayer) {
        Vector newDelta = Vector(layer.outSize);
        for (int i = 0; i < layer.outSize; i++) {
            DTYPE coreGradient = 0;
            for (int j = 0; j < delta.n; j++) {
                coreGradient += delta[j] * derivatives[i] * previousWeights[j][i];
            }

            layer.biases[i] -= learningRate * coreGradient;

            for (int j = 0; j < layer.inSize; j++) {
                layer.weights[i][j] -= learningRate * coreGradient * layer.data[j];
            }

            newDelta[i] = coreGradient;
        }

        return newDelta;
    } else {
        Vector newDelta = delta;
        for (int i = 0; i < layer.outSize; i++) {
            DTYPE coreGradient = delta[i] * derivatives[i];

            layer.biases[i] -= learningRate * coreGradient;

            for (int j = 0; j < layer.inSize; j++) {
                layer.weights[i][j] -= learningRate * coreGradient * layer.data[j];
            }

            newDelta.data[i] = coreGradient;
        }
        return newDelta;
    }
}

#endif