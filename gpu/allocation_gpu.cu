//
// Created by Jan Warchocki on 10/03/2022.
//

#include "allocation_gpu.cuh"
#include "verify.cuh"
#include "assert.cuh"
#include <stdio.h>

#ifdef HAS_CUDA

DTYPE* allocate1DArrayDevice(int n) {
    DTYPE* allocated;
    gpuCheckError( hipMalloc(&allocated, n * sizeof(DTYPE)) );
    return allocated;
}

void copy1DFromHostToDevice(DTYPE* host, DTYPE* device, int n) {
    gpuCheckError( hipMemcpy(device, host, n * sizeof(DTYPE), hipMemcpyHostToDevice) );
}

void copy2DFromHostToDevice(DTYPE** host, DTYPE* device, int n, int m) {
    DTYPE* temp = allocate1DArray(n * m);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            temp[i * m + j] = host[i][j];
        }
    }
    gpuCheckError( hipMemcpy(device, temp, n * m * sizeof(DTYPE), hipMemcpyHostToDevice) );
    free(temp);
}

void free1DArrayDevice(DTYPE* device) {
    gpuCheckError( hipFree(device) );
}

void copy1DFromDeviceToHost(DTYPE* device, DTYPE* host, int n) {
    gpuCheckError( hipMemcpy(host, device, n * sizeof(DTYPE), hipMemcpyDeviceToHost) );
}

void copy2DFromDeviceToHost(DTYPE* device, DTYPE** host, int n, int m) {
    DTYPE* temp = allocate1DArray(n * m);
    gpuCheckError( hipMemcpy(temp, device, n * m * sizeof(DTYPE), hipMemcpyDeviceToHost) );

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            DTYPE val = temp[i * m + j];
            host[i][j] = val;
        }
    }

    free(temp);
}

#else

DTYPE* allocate1DArrayDevice(int n) {
    return nullptr;
}

DTYPE** allocate2DArrayDevice(int n, int m) {
    return nullptr;
}

void copy1DFromHostToDevice(DTYPE* host, DTYPE* device, int n) {

}

void copy2DFromHostToDevice(DTYPE** host, DTYPE** device, int n, int m) {

}

#endif