#include "hip/hip_runtime.h"
//
// Created by Jan Warchocki on 29/05/2022.
//

#include "../gpu/allocation_gpu.cuh"
#include "../gpu/assert.cuh"
#include "backpropagation.h"
#include "verify.cuh"
#include <exceptions/unexpected_cuda_call_exception.h>

#ifdef HAS_CUDA

//NOLINTBEGIN(readability-static-accessed-through-instance)

__global__ void applyGradientsKernel(DTYPE* biases, DTYPE* weights, DTYPE* biasesGradients, DTYPE* weightsGradients,
                                     size_t inSize, size_t outSize, size_t batchSize, DTYPE learningRate) {
    auto outIndex = blockIdx.x;
    auto inIndex = threadIdx.x;

    if (outIndex >= outSize || inIndex >= inSize) {
        return;
    }

    if (inIndex == 0) {
        biases[outIndex] -= learningRate * biasesGradients[outIndex] / static_cast<DTYPE>(batchSize);
        biasesGradients[outIndex] = 0;
    }

    weights[inIndex * outSize + outIndex] -=
            learningRate * weightsGradients[inIndex * outSize + outIndex] / static_cast<DTYPE>(batchSize);
    weightsGradients[inIndex * outSize + outIndex] = 0;
}

//NOLINTEND(readability-static-accessed-through-instance)

void applyGradientsOnDevice(Layer& layer, size_t batchSize, DTYPE learningRate) {
    applyGradientsKernel<<<layer.outSize, layer.inSize>>>(layer.biases.data, layer.weights.data,
                                                          layer.biasesGradients.data, layer.weightsGradients.data,
                                                          layer.inSize, layer.outSize, batchSize, learningRate);
    GPU_CHECK_ERROR(hipGetLastError());
    GPU_CHECK_ERROR(hipDeviceSynchronize());
}

#else

void applyGradientsOnDevice(Layer& layer, size_t batchSize, DTYPE learningRate) {
    throw UnexpectedCUDACallException();
}

#endif