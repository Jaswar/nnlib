/**
 * @file session.cpp
 * @brief Source file defining the Session class.
 * @author Jan Warchocki
 * @date 29 August 2022
 */

#include "session.cuh"
#include "verify.cuh"

Session::Session() {
    numCores = std::thread::hardware_concurrency();
#ifdef HAS_CUDA
    // NOLINTNEXTLINE(cppcoreguidelines-pro-type-member-init)
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    threadsPerBlock = props.maxThreadsPerBlock;
#else
    threadsPerBlock = 0;
#endif
}
