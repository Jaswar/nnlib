//
// Created by Jan Warchocki on 10/03/2022.
//

#include "allocation_gpu.cuh"
#include "assert.cuh"
#include "verify.cuh"
#include <exceptions/unexpected_cuda_call_exception.h>

#ifdef HAS_CUDA

DTYPE* allocate1DArrayDevice(size_t n) {
    DTYPE* allocated;
    gpuCheckError(hipMalloc(&allocated, n * sizeof(DTYPE)));
    return allocated;
}

void copy1DFromDeviceToDevice(DTYPE* oldLoc, DTYPE* newLoc, size_t n) {
    gpuCheckError(hipMemcpy(newLoc, oldLoc, n * sizeof(DTYPE), hipMemcpyDeviceToDevice));
}

void copy1DFromHostToDevice(DTYPE* host, DTYPE* device, size_t n) {
    gpuCheckError(hipMemcpy(device, host, n * sizeof(DTYPE), hipMemcpyHostToDevice));
}

void copy2DFromHostToDevice(DTYPE** host, DTYPE* device, size_t n, size_t m) {
    DTYPE* temp = allocate1DArray(n * m);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            temp[i * m + j] = host[i][j];
        }
    }
    gpuCheckError(hipMemcpy(device, temp, n * m * sizeof(DTYPE), hipMemcpyHostToDevice));
    free(temp);
}

void free1DArrayDevice(DTYPE* device) {
    gpuCheckError(hipFree(device));
}

void copy1DFromDeviceToHost(DTYPE* device, DTYPE* host, size_t n) {
    gpuCheckError(hipMemcpy(host, device, n * sizeof(DTYPE), hipMemcpyDeviceToHost));
}

void copy2DFromDeviceToHost(DTYPE* device, DTYPE** host, size_t n, size_t m) {
    DTYPE* temp = allocate1DArray(n * m);
    gpuCheckError(hipMemcpy(temp, device, n * m * sizeof(DTYPE), hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            DTYPE val = temp[i * m + j];
            host[i][j] = val;
        }
    }

    free(temp);
}

DTYPE* copy1DArrayDevice(size_t n, DTYPE* old) {
    DTYPE* allocated = allocate1DArrayDevice(n);
    gpuCheckError(hipMemcpy(allocated, old, n * sizeof(DTYPE), hipMemcpyDeviceToDevice));
    return allocated;
}

#else

DTYPE* allocate1DArrayDevice(size_t n) {
    throw UnexpectedCUDACallException();
}

void copy1DFromDeviceToDevice(DTYPE* oldLoc, DTYPE* newLoc, size_t n) {
    throw UnexpectedCUDACallException();
}

void copy1DFromHostToDevice(DTYPE* host, DTYPE* device, size_t n) {
    throw UnexpectedCUDACallException();
}

void copy2DFromHostToDevice(DTYPE** host, DTYPE* device, size_t n, size_t m) {
    throw UnexpectedCUDACallException();
}

void free1DArrayDevice(DTYPE* device) {
    throw UnexpectedCUDACallException();
}

void copy1DFromDeviceToHost(DTYPE* device, DTYPE* host, size_t n) {
    throw UnexpectedCUDACallException();
}

void copy2DFromDeviceToHost(DTYPE* device, DTYPE** host, size_t n, size_t m) {
    throw UnexpectedCUDACallException();
}

DTYPE* copy1DArrayDevice(size_t n, DTYPE* old) {
    throw UnexpectedCUDACallException();
}

#endif