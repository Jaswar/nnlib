/**
 * @file allocation_gpu.cu
 * @brief Source file defining methods regarding memory allocation on device.
 * @author Jan Warchocki
 * @date 10 March 2022
 */

#include "allocation_gpu.cuh"
#include "assert.cuh"
#include "verify.cuh"
#include <exceptions/unexpected_cuda_call_exception.h>

#ifdef __CUDA__

float* allocate1DArrayDevice(size_t n) {
    float* allocated;
    GPU_CHECK_ERROR(hipMalloc(&allocated, n * sizeof(float)));
    return allocated;
}

void copy1DFromDeviceToDevice(float* oldLoc, float* newLoc, size_t n) {
    GPU_CHECK_ERROR(hipMemcpy(newLoc, oldLoc, n * sizeof(float), hipMemcpyDeviceToDevice));
}

void copy1DFromHostToDevice(float* host, float* device, size_t n) {
    GPU_CHECK_ERROR(hipMemcpy(device, host, n * sizeof(float), hipMemcpyHostToDevice));
}

void copy2DFromHostToDevice(float** host, float* device, size_t n, size_t m) {
    float* temp = allocate1DArray(n * m);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            temp[i * m + j] = host[i][j];
        }
    }
    GPU_CHECK_ERROR(hipMemcpy(device, temp, n * m * sizeof(float), hipMemcpyHostToDevice));
    free(temp);
}

void free1DArrayDevice(float* device) {
    GPU_CHECK_ERROR(hipFree(device));
}

void copy1DFromDeviceToHost(float* device, float* host, size_t n) {
    GPU_CHECK_ERROR(hipMemcpy(host, device, n * sizeof(float), hipMemcpyDeviceToHost));
}

void copy2DFromDeviceToHost(float* device, float** host, size_t n, size_t m) {
    float* temp = allocate1DArray(n * m);
    GPU_CHECK_ERROR(hipMemcpy(temp, device, n * m * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            float val = temp[i * m + j];
            host[i][j] = val;
        }
    }

    free(temp);
}

float* copy1DArrayDevice(size_t n, float* old) {
    float* allocated = allocate1DArrayDevice(n);
    GPU_CHECK_ERROR(hipMemcpy(allocated, old, n * sizeof(float), hipMemcpyDeviceToDevice));
    return allocated;
}

#else

float* allocate1DArrayDevice(size_t n) {
    throw UnexpectedCUDACallException();
}

void copy1DFromDeviceToDevice(float* oldLoc, float* newLoc, size_t n) {
    throw UnexpectedCUDACallException();
}

void copy1DFromHostToDevice(float* host, float* device, size_t n) {
    throw UnexpectedCUDACallException();
}

void copy2DFromHostToDevice(float** host, float* device, size_t n, size_t m) {
    throw UnexpectedCUDACallException();
}

void free1DArrayDevice(float* device) {
    throw UnexpectedCUDACallException();
}

void copy1DFromDeviceToHost(float* device, float* host, size_t n) {
    throw UnexpectedCUDACallException();
}

void copy2DFromDeviceToHost(float* device, float** host, size_t n, size_t m) {
    throw UnexpectedCUDACallException();
}

float* copy1DArrayDevice(size_t n, float* old) {
    throw UnexpectedCUDACallException();
}

#endif