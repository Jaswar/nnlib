#include "hip/hip_runtime.h"
//
// Created by Jan Warchocki on 10/03/2022.
//

#include "backpropagation.cuh"
#include "../gpu/verify.cuh"
#include "../gpu/allocation_gpu.cuh"
#include "../gpu/assert.cuh"

#ifdef HAS_CUDA

__global__
void performBackpropagation(DTYPE* biases, DTYPE* weights, DTYPE* data, DTYPE* derivatives,
                            DTYPE* delta, DTYPE* previousWeights, DTYPE* newDelta, int inSize, int outSize, int deltaSize,
                            DTYPE learningRate, bool isLastLayer) {
    auto index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= outSize) {
        return;
    }

    if (isLastLayer) {
        DTYPE coreGradient = delta[index] * derivatives[index];

        biases[index] -= learningRate * coreGradient;

        for (int j = 0; j < inSize; j++) {
            weights[index * inSize + j] -= learningRate * coreGradient * data[j];
        }

        newDelta[index] = coreGradient;
    } else {
        DTYPE coreGradient = 0;
        for (int j = 0; j < deltaSize; j++) {
            coreGradient += delta[j] * derivatives[index] * previousWeights[j * outSize + index];
        }

        biases[index] -= learningRate * coreGradient;

        for (int j = 0; j < inSize; j++) {
            weights[index * inSize + j] -= learningRate * coreGradient * data[j];
        }

        newDelta[index] = coreGradient;
    }
}

void backpropagation(Layer& layer, const Vector& delta, const Matrix& previousWeights,
              bool isLastLayer, DTYPE learningRate) {
    performBackpropagation<<<1, layer.outSize>>>(layer.biases.data, layer.weights.data,
                                                 layer.data->data, layer.derivatives.data, delta.data,
                                                 previousWeights.data, layer.newDelta.data,
                                                 layer.inSize, layer.outSize, delta.n, learningRate, isLastLayer);
}

#else

Vector backpropagation(Layer& layer, const Vector& delta, const Matrix& previousWeights,
              bool isLastLayer, DTYPE learningRate) {
    const Vector& derivatives = layer.calculateDerivatives();
    if (!isLastLayer) {
        Vector newDelta = Vector(layer.outSize);
        for (int i = 0; i < layer.outSize; i++) {
            DTYPE coreGradient = 0;
            for (int j = 0; j < delta.n; j++) {
                coreGradient += delta[j] * derivatives[i] * previousWeights[j][i];
            }

            layer.biases[i] -= learningRate * coreGradient;

            for (int j = 0; j < layer.inSize; j++) {
                layer.weights[i][j] -= learningRate * coreGradient * layer.data[j];
            }

            newDelta[i] = coreGradient;
        }

        return newDelta;
    } else {
        Vector newDelta = delta;
        for (int i = 0; i < layer.outSize; i++) {
            DTYPE coreGradient = delta[i] * derivatives[i];

            layer.biases[i] -= learningRate * coreGradient;

            for (int j = 0; j < layer.inSize; j++) {
                layer.weights[i][j] -= learningRate * coreGradient * layer.data[j];
            }

            newDelta.data[i] = coreGradient;
        }
        return newDelta;
    }
}

#endif